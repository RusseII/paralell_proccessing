#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed:_____________________________________ Date:_____________        */
/*                                                                        */
/*                                                                        */
/* 3460:4/577 CUDA Vector Add lab, Version 1.01, Fall 2016.               */

#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
	int i=threadIdx.x+(blockIdx.x*blockDim); //finds thread ID
	out[i]=(in1[1]+in2[i];
}

int main(int argc, char **argv) {
  int inputLength1, inputLength2, outputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
  float *expectedOutput;

  FILE *infile1, *infile2, *outfile;
  unsigned int generic, gpu, copy, compute, blog = 1;

  // Import host input data
  CUT_SAFE_CALL(cutCreateTimer(&generic));
  cutStartTimer(generic);
  if ((infile1 = fopen("input0.raw", "r")) == NULL)
  { printf("Cannot open input0.raw.\n"); exit(EXIT_FAILURE); }
  if ((infile2 = fopen("input1.raw", "r")) == NULL)
  { printf("Cannot open input1.raw.\n"); exit(EXIT_FAILURE); }
  fscanf(infile1, "%i", &inputLength1);
  hostInput1 = (float*) malloc(sizeof(float) * inputLength1);
  for (int i = 0; i < inputLength1; i++)
    fscanf(infile1, "%f", &hostInput1[i]);
  fscanf(infile2, "%i", &inputLength2);
  hostInput2 = (float*) malloc(sizeof(float) * inputLength2);
  for (int i = 0; i < inputLength2; i++)
    fscanf(infile2, "%f", &hostInput2[i]);
  fclose(infile1);
  fclose(infile2);
  hostOutput = (float *)malloc(sizeof(float) * inputLength1);
  cutStopTimer(generic);
  printf("Importing data and creating memory on host: %f ms\n", cutGetTimerValue(generic));

  if (blog) printf("*** The input length is %i\n", inputLength1);

  CUT_SAFE_CALL(cutCreateTimer(&gpu));
  cutStartTimer(gpu);
  //@@ Allocate GPU memory here
int alloSize=sizeof(float)*inputLength1;
hipMalloc((void * *) & deviceInput1,alloSize);
hipMalloc((void * *) & deviceInput2,alloSize);
hipMalloc((void * *) & deviceOutput,alloSize);
  
  cutStopTimer(gpu);
  printf("Allocating GPU memory: %f ms\n", cutGetTimerValue(gpu));

  cutDeleteTimer(gpu);
  CUT_SAFE_CALL(cutCreateTimer(&gpu));
  cutStartTimer(gpu);

  //@@ Copy memory to the GPU here
hipMemcpy(deviceInput1,hostInput1,alloSize,hipMemcpyHostToDevice)
hipMemcpy(deviceInput2,hostInput1,alloSize,hipMemcpyHostToDevice)
hipMemcpy(deviceOutput,hostInput1,alloSize,hipMemcpyHostToDevice)
  cutStopTimer(gpu);
  printf("Copying input memory to the GPU: %f ms\n", cutGetTimerValue(gpu));

  //@@ Initialize the grid and block dimensions here
dim3 gridDim(ceil(inputLength1/4),1,1);
dim3 blockDim(ceil(inputLength1/4),1,1);



  if (blog) printf("*** Block dimension is %i\n", blockDim.x);
  if (blog) printf("*** Grid dimension is %i\n", gridDim.x);

  CUT_SAFE_CALL(cutCreateTimer(&compute));
  cutStartTimer(compute);
  
  //@@ Launch the GPU Kernel here
  vecAdd <<< gridDim, blockDim >>> (deviceInput1,deviceInput2, deviceOutput,inputLength1);

  hipDeviceSynchronize();
  
  cutStopTimer(compute);
  printf("Performing CUDA computation: %f ms\n", cutGetTimerValue(compute));

  CUT_SAFE_CALL(cutCreateTimer(&copy));
  cutStartTimer(copy);

  //@@ Copy the GPU memory back to the CPU here
hipMemcpy(hostOutput, deviceOutput, alloSize, hipMemcpyDeviceToHost);

  
  cutStopTimer(copy);
  printf("Copying output memory to the CPU: %f ms\n", cutGetTimerValue(copy));

  cutDeleteTimer(gpu);
  CUT_SAFE_CALL(cutCreateTimer(&gpu));
  cutStartTimer(gpu);
  
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  cutStopTimer(gpu);
  printf("Freeing GPU Memory: %f ms\n", cutGetTimerValue(gpu));

  if ((outfile = fopen("output.raw", "r")) == NULL)
  { printf("Cannot open output.raw.\n"); exit(EXIT_FAILURE); }
  fscanf(outfile, "%i", &outputLength);
  expectedOutput = (float*) malloc(sizeof(float) * outputLength);
  for (int i = 0; i < outputLength; i++)
    fscanf(outfile, "%f", &expectedOutput[i]);
  fclose(outfile);
  int test = 1;
  for (int i = 0; i < outputLength; i++)
    test = test && (abs(expectedOutput[i] - hostOutput[i]) < 0.005);
  if (test) printf("Results correct.\n");
  else printf("Results incorrect.\n");

  cutDeleteTimer(generic);
  cutDeleteTimer(gpu);
  cutDeleteTimer(copy);
  cutDeleteTimer(compute);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(expectedOutput);
  
  return 0;
}
